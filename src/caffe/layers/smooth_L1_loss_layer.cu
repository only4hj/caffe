#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Fast R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Ross Girshick
// ------------------------------------------------------------------

#include "caffe/fast_rcnn_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SmoothL1Forward(const int n, const Dtype* in, Dtype* out) {
  // f(x) = 0.5 * x^2    if |x| < 1
  //        |x| - 0.5    otherwise
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = in[index];
    Dtype abs_val = abs(val);
    if (abs_val < 1) {
      out[index] = 0.5 * val * val;
    } else {
      out[index] = abs_val - 0.5;
    }
  }
}

template <typename Dtype>
void SmoothL1LossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();


  //printf ("SmoothL1LossLayer::Forward_gpu()\n");
  Blob<Dtype>* bbox_targets = bottom[1];
  Blob<Dtype>* bbox_loss_weights = bottom[2];
  Blob<Dtype>* labels = bottom[4];

  int DEBUG = 0;

  if (DEBUG == 1) {
	  int label_pos_no = 0;
	  int label_zero_no = 0;
	  int label_neg_no = 0;
	  int bbox_target_zero_no = 0;
	  int bbox_target_non_zero_no = 0;
	  int bbox_loss_weight_pos_no = 0;
	  int bbox_loss_weight_non_pos_no = 0;
  
	  for (int i = 0; i<2; i++)
		for (int j=0; j<36; j++)
		  for (int k=0; k<bbox_targets->height(); k++)
			  for (int l=0; l<bbox_targets->width(); l++) {
				Dtype bbox_target = bbox_targets->data_at(i, j, k, l);
				Dtype bbox_loss_weight = bbox_loss_weights->data_at(i, j, k, l);
			
				if (bbox_target != 0)
					bbox_target_non_zero_no++;
				else
					bbox_target_zero_no++;

				if (bbox_loss_weight > 0)
					bbox_loss_weight_pos_no++;
				else
					bbox_loss_weight_non_pos_no++;
			}

	  int value_match_no = 0;
  
	  for (int i = 0; i<2; i++)
	    for (int j=0; j<9; j++)
		  for (int k=0; k<bbox_targets->height(); k++)
			for (int l=0; l<bbox_targets->width(); l++) {
				Dtype label = labels->data_at(i, j, k, l);
			
				if (label == 1)
					label_pos_no++;
				else if (label == 0)
					label_zero_no++;
				else if (label < 0)
					label_neg_no++;

				Dtype bbox_target1 = bbox_targets->data_at(i, j*4, k, l);
				Dtype bbox_target2 = bbox_targets->data_at(i, j*4+1, k, l);
				Dtype bbox_target3 = bbox_targets->data_at(i, j*4+2, k, l);
				Dtype bbox_target4 = bbox_targets->data_at(i, j*4+3, k, l);
				Dtype bbox_loss_weight1 = bbox_loss_weights->data_at(i, j*4, k, l);
				Dtype bbox_loss_weight2 = bbox_loss_weights->data_at(i, j*4+1, k, l);
				Dtype bbox_loss_weight3 = bbox_loss_weights->data_at(i, j*4+2, k, l);
				Dtype bbox_loss_weight4 = bbox_loss_weights->data_at(i, j*4+3, k, l);

				if (label == 1){
					if (bbox_target1 != 0 && bbox_target2 != 0 && bbox_target3 != 0 && bbox_target4 != 0 && 
						bbox_loss_weight1 == 1 && bbox_loss_weight2 == 1 && bbox_loss_weight3 == 1 && bbox_loss_weight4 == 1)
						value_match_no++;
				}
			}

	  printf ("label_pos_no : %d\n", label_pos_no);
	  printf ("label_zero_no : %d\n", label_zero_no);
	  printf ("label_neg_no : %d\n", label_neg_no);
	  printf ("bbox_target_non_zero_no : %d\n", bbox_target_non_zero_no);
	  printf ("bbox_target_zero_no : %d\n", bbox_target_zero_no);
	  printf ("bbox_loss_weight_non_pos_no : %d\n", bbox_loss_weight_non_pos_no);
	  printf ("bbox_loss_weight_pos_no : %d\n", bbox_loss_weight_pos_no);
	  printf ("value_match_no : %d\n", value_match_no);
	  printf ("SmoothL1LossLayer count : %d\n", count);
	  printf ("batch_size_ : %d\n", batch_size_);
  }


  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());    // d := b0 - b1
  if (has_weights_) {
    caffe_gpu_mul(
        count,
        bottom[2]->gpu_data(),
        diff_.gpu_data(),
        diff_.mutable_gpu_data());  // d := w * (b0 - b1)
  }
  SmoothL1Forward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, diff_.gpu_data(), errors_.mutable_gpu_data());
  CUDA_POST_KERNEL_CHECK;

  Dtype loss;
  caffe_gpu_asum(count, errors_.gpu_data(), &loss);
  
  if (batch_size_ > 0)
    top[0]->mutable_cpu_data()[0] = loss / batch_size_;
  else
    top[0]->mutable_cpu_data()[0] = loss / bottom[0]->num();
}

template <typename Dtype>
__global__ void SmoothL1Backward(const int n, const Dtype* in, Dtype* out) {
  // f'(x) = x         if |x| < 1
  //       = sign(x)   otherwise
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = in[index];
    Dtype abs_val = abs(val);
    if (abs_val < 1) {
      out[index] = val;
    } else {
      out[index] = (Dtype(0) < val) - (val < Dtype(0));
    }
  }
}

template <typename Dtype>
void SmoothL1LossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  int count = diff_.count();
  SmoothL1Backward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, diff_.gpu_data(), diff_.mutable_gpu_data());
  CUDA_POST_KERNEL_CHECK;
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      
	  // DJDJ
	  Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();

	  if (batch_size_ > 0)
		alpha = sign * top[0]->cpu_diff()[0] / batch_size_;

      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                           // alpha
          diff_.gpu_data(),                // x
          Dtype(0),                        // beta
          bottom[i]->mutable_gpu_diff());  // y
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SmoothL1LossLayer);

}  // namespace caffe
