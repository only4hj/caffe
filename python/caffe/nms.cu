#include "hip/hip_runtime.h"
#include <Windows.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <algorithm>

#pragma comment(lib, "cudaRT.lib")
#pragma comment(lib, "winmm.lib")

using namespace std;

std::vector<int> nms_cpp(float* x1s, float* y1s, float* x2s, float* y2s, float* scores,
	int data_size, float thresh, int max_candidate);

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ VOID SuppressFunc(float ix1, float iy1, float ix2, float iy2, 
			float* x1s, float* y1s, float* x2s, float* y2s, 
			int iArea, int base_index, int data_size, int* suppressed,
			float thresh) {

	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	int j = base_index + threadID;

	//printf ("data_size : %d\n", data_size);
	//printf ("base_index : %d, blockIdx.x : %d, blockDim.x : %d, threadIdx.x : %d, j : %d\n", base_index, blockIdx.x, blockDim.x, threadIdx.x, j);
	
	if (j < data_size) {
		//printf ("suppressed[j] : %d\n", suppressed[j]);
		if (suppressed[j] == 1)
			return;
		int jArea = (x2s[j] - x1s[j] + 1) * (y2s[j] - y1s[j] + 1);
		float xx1 = max(ix1, x1s[j]);
		float yy1 = max(iy1, y1s[j]);
		float xx2 = min(ix2, x2s[j]);
		float yy2 = min(iy2, y2s[j]);
		float w = max(0.0, xx2 - xx1 + 1);
		float h = max(0.0, yy2 - yy1 + 1);
		float inter = w * h;
		float ovr = (float)inter / float(iArea + jArea - inter);
		if (ovr >= thresh) {
			suppressed[j] = 1;
			//printf("%d is suppressed by %d\n", j, base_index-1);
		}
	}
}

std::vector<int> nms_cuda(float* x1s, float* y1s, float* x2s, float* y2s, float* scores,
		int data_size, float thresh, int max_candidate) {

	std::vector<int> keep;
	int keep_no = 0;
	int suppressed_size = sizeof(int) * data_size;
	int* suppressed = (int*)malloc(suppressed_size);
	memset(suppressed, 0, sizeof(int)*data_size);
	
	int points_size = data_size * sizeof(float);
	float* d_x1s, *d_y1s, *d_x2s, *d_y2s;
	int* d_suppressed;
	hipMalloc((void **)&d_x1s, points_size);
	hipMalloc((void **)&d_y1s, points_size);
	hipMalloc((void **)&d_x2s, points_size);
	hipMalloc((void **)&d_y2s, points_size);
	hipMalloc((void **)&d_suppressed, sizeof(int)*data_size);

	hipMemcpy(d_x1s, x1s, points_size, hipMemcpyHostToDevice);
	hipMemcpy(d_y1s, y1s, points_size, hipMemcpyHostToDevice);
	hipMemcpy(d_x2s, x2s, points_size, hipMemcpyHostToDevice);
	hipMemcpy(d_y2s, y2s, points_size, hipMemcpyHostToDevice);
	
	for (int _i = 0; _i < data_size; _i++){
		int i = _i;

		if (suppressed[i] == 1)
			continue;

		keep.push_back(i);
		keep_no++;

		if (keep_no == max_candidate)
			break;

		float ix1 = x1s[i];
		float iy1 = y1s[i];
		float ix2 = x2s[i];
		float iy2 = y2s[i];
		int iArea = (ix2 - ix1 + 1) * (iy2 - iy1 + 1);

		hipMemcpy(d_suppressed, suppressed, suppressed_size, hipMemcpyHostToDevice);

		int block_size = 100;
		int thread_size = data_size / block_size - (i / block_size);
		if (data_size % block_size != 0)
			thread_size++;		 
		SuppressFunc <<<block_size, thread_size >>>(ix1, iy1, ix2, iy2, 
							d_x1s, d_y1s, d_x2s, d_y2s, 
							iArea, i+1, data_size, d_suppressed,
							thresh);

		hipMemcpy(suppressed, d_suppressed, suppressed_size, hipMemcpyDeviceToHost);
		
		//hipDeviceSynchronize();
		//gpuErrchk( hipPeekAtLastError() );
	}
	
	free(suppressed);

	hipFree(d_x1s);
	hipFree(d_y1s);
	hipFree(d_x2s);
	hipFree(d_y2s);
	hipFree(d_suppressed);

	return keep;
}

int nms_cu_main(void) {
	DWORD dwTime = 0;
	float thresh = 0.2;
	int max_candidate = 123;
	std::vector<float> x1s;
	std::vector<float> y1s;
	std::vector<float> x2s;
	std::vector<float> y2s;
	std::vector<float> scores;

	for (int i = 0; i < 20000; i++) {
		float x1 = rand() % 100;
		float y1 = rand() % 100;
		float x2 = x1 + rand() % 500;
		float y2 = y1 + rand() % 500;
		float score = (float)(rand() % 100) / (float)100;;
		x1s.push_back(x1);
		y1s.push_back(y1);
		x2s.push_back(x2);
		y2s.push_back(y2);
		scores.push_back(score);
	}

	int data_size = x1s.size();

	std::cout << "input rect : " << data_size << std::endl;

	for (int i=0; i<50; i++) {
		dwTime = timeGetTime();

		std::vector<int> keep1 = nms_cpp(x1s.data(), y1s.data(), x2s.data(), y2s.data(), scores.data(), 
						data_size, thresh, max_candidate);

		dwTime = timeGetTime() - dwTime;
		std::cout << "keep size  : " << keep1.size() << std::endl;
		std::cout << "time " << dwTime << "ms." << std::endl;


		dwTime = timeGetTime();

		std::vector<int> keep2 = nms_cuda(x1s.data(), y1s.data(), x2s.data(), y2s.data(), scores.data(), 
						data_size, thresh, max_candidate);

		dwTime = timeGetTime() - dwTime;
		std::cout << "keep2 size  : " << keep2.size() << std::endl;
		std::cout << "time " << dwTime << "ms." << std::endl << std::endl;
	}

	//for (int i = 0; i < rect_size; i++)
	//	std::cout << "suppressed[" << i << "] : " << suppressed[i] << std::endl;

	//for (int i = 0; i < keep.size(); i++)
	//	std::cout << "keep[" << i << "] : " << keep[i] << std::endl;



	return 0;
}
